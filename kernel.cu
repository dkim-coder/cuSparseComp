#include <iostream>
#include <hip/hip_runtime_api.h>      // hipMalloc, hipMemcpy, etc.
#include <cusparseLt.h>            // cusparseLt header  --> #include <hipsparse.h>

                                                                 
#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}                                                                               

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}  


constexpr int EXIT_UNSUPPORTED = 2;

// cusparseLtMatmul  -->  D = alpha * A * B + beta * C
template <typename T>
int cusLtMatmul(T* hA, T* hB, T* hC, T* hA_pruned, const int m, const int n, const int k, const int device_num = 0)
{
    int major_cc, minor_cc;
    CHECK_CUDA(hipDeviceGetAttribute(&major_cc, hipDeviceAttributeComputeCapabilityMajor, device_num))
    CHECK_CUDA(hipDeviceGetAttribute(&minor_cc, hipDeviceAttributeComputeCapabilityMinor, device_num))
    if (!(major_cc == 8 && minor_cc == 0) && !(major_cc == 8 && minor_cc == 6)) {
        std::cout << "\ncusparseLt is supported only on GPU devices with compute capability == 8.0, 8.6 current: " << major_cc << "." << minor_cc << std::endl;
        return EXIT_UNSUPPORTED;
    }
    
    // matrix A, B, C : NON_TRANSPOSE, row-major
    auto          order = HIPSPARSE_ORDER_ROW;
    auto          opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          type = HIP_R_32F;
    auto          compute_type = CUSPARSE_COMPUTE_TF32;
    bool     is_rowmajor = (order == HIPSPARSE_ORDER_ROW);
    bool     isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    bool     isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    auto     num_A_rows = (isA_transposed) ? k : m;
    auto     num_A_cols = (isA_transposed) ? m : k;
    auto     num_B_rows = (isB_transposed) ? n : k;
    auto     num_B_cols = (isB_transposed) ? k : n;
    auto     num_C_rows = m;
    auto     num_C_cols = n;
    unsigned alignment = 32;    // mutiple of 32   (processor�� 8byte������ �о�� 64bit os) --> ���μ��� ���� Ƚ�� �ٿ���
    auto     lda = (is_rowmajor) ? num_A_cols : num_A_rows;
    auto     ldb = (is_rowmajor) ? num_B_cols : num_B_rows;
    auto     ldc = (is_rowmajor) ? num_C_cols : num_C_rows;
    auto     A_height = (is_rowmajor) ? num_A_rows : num_A_cols;
    auto     B_height = (is_rowmajor) ? num_B_rows : num_B_cols;
    auto     C_height = (is_rowmajor) ? num_C_rows : num_C_cols;
    auto     A_size = A_height * lda * sizeof(T);
    auto     B_size = B_height * ldb * sizeof(T);
    auto     C_size = C_height * ldc * sizeof(T);

    float alpha = 1.0f;
    float beta = 0.0f;

    // Device memory management
    T* dA, * dB, * dC, * dD, * dA_compressed;
    CHECK_CUDA(hipMalloc((void**)&dA, A_size))  
    CHECK_CUDA(hipMalloc((void**)&dB, B_size))  
    CHECK_CUDA(hipMalloc((void**)&dC, C_size))  
    dD = dC;

    CHECK_CUDA(hipMemcpy(dA, hA, A_size, hipMemcpyHostToDevice))  
    CHECK_CUDA(hipMemcpy(dB, hB, B_size, hipMemcpyHostToDevice))  
    CHECK_CUDA(hipMemset(dC, 0, C_size))   

    // ---------------------------------------------------------------------------------------------
    cusparseLtHandle_t             handle;  
    cusparseLtMatDescriptor_t      matA, matB, matC;    
    cusparseLtMatmulDescriptor_t   matmul;  
    cusparseLtMatmulAlgSelection_t alg_sel; 
    cusparseLtMatmulPlan_t         plan;     
    hipStream_t                   stream = nullptr;
    
    CHECK_CUSPARSE(cusparseLtInit(&handle))
    // matrix descriptor initialization
    CHECK_CUSPARSE(cusparseLtStructuredDescriptorInit(&handle, &matA, num_A_rows, num_A_cols, lda, alignment, type, order, CUSPARSELT_SPARSITY_50_PERCENT)) 
    CHECK_CUSPARSE(cusparseLtDenseDescriptorInit(&handle, &matB, num_B_rows, num_B_cols, ldb, alignment, type, order))  
    CHECK_CUSPARSE(cusparseLtDenseDescriptorInit(&handle, &matC, num_C_rows, num_C_cols, ldc, alignment, type, order))  
    // matmul, algorithm selection, and plan initialization
    CHECK_CUSPARSE(cusparseLtMatmulDescriptorInit(&handle, &matmul, opA, opB, &matA, &matB, &matC, &matC, compute_type))    
    CHECK_CUSPARSE(cusparseLtMatmulAlgSelectionInit(&handle, &alg_sel, &matmul, CUSPARSELT_MATMUL_ALG_DEFAULT)) 
    int alg = 0;    // algorithm attribute number
    CHECK_CUSPARSE(cusparseLtMatmulAlgSetAttribute(&handle, &alg_sel, CUSPARSELT_MATMUL_ALG_CONFIG_ID, &alg, sizeof(alg)))   
    size_t workspace_size;
    workspace_size = 10000;
    //CHECK_CUSPARSE(cusparseLtMatmulGetWorkspace(&handle, &plan, &workspace_size))
    CHECK_CUSPARSE(cusparseLtMatmulPlanInit(&handle, &plan, &matmul, &alg_sel, workspace_size)) 
    // Prune the A matrix (in-place) and check the correcteness
    int* d_valid;
    CHECK_CUDA(hipMalloc((void**)&d_valid, sizeof(d_valid)))
    CHECK_CUSPARSE(cusparseLtSpMMAPrune(&handle, &matmul, dA, dA, CUSPARSELT_PRUNE_SPMMA_TILE, stream)) 
    CHECK_CUSPARSE(cusparseLtSpMMAPruneCheck(&handle, &matmul, dA, d_valid, stream))    
    int is_valid;
    CHECK_CUDA(hipMemcpyAsync(&is_valid, d_valid, sizeof(*d_valid), hipMemcpyDeviceToHost, stream))
    CHECK_CUDA(hipStreamSynchronize(stream));
    if (is_valid != 0) {
        std::cout << "!!!! The matrix has been pruned in a wrong way. cusparseLtMatmul will not provide correct results" << std::endl;
        return EXIT_FAILURE;
    }

    size_t compressed_size;
    CHECK_CUSPARSE(cusparseLtSpMMACompressedSize(&handle, &plan, &compressed_size)) 
    CHECK_CUDA(hipMalloc((void**)&dA_compressed, compressed_size)) // memory allocation dA_compressed
    CHECK_CUSPARSE(cusparseLtSpMMACompress(&handle, &plan, dA, dA_compressed, stream))  

    
    void* d_workspace = nullptr;
    int           num_streams = 0;
    hipStream_t* streams = nullptr;
    /* CHECK_CUSPARSE(cusparseLtMatmulSearch(&handle, &plan, &alpha, dA_compressed, dB, &beta, dC, dD, d_workspace, streams, num_streams))   // ------> ���� �κ�
         int alg_id;
         CHECK_CUSPARSE(cusparseLtMatmulAlgGetAttribute(&handle, &alg_sel, CUSPARSELT_MATMUL_ALG_CONFIG_ID, &alg_id, sizeof(alg_id)))
         std::cout << "alg_id : " << alg_id << std::endl; */

    // Perform the matrix multiplication
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    CHECK_CUSPARSE(cusparseLtMatmul(&handle, &plan, &alpha, dA_compressed, dB, &beta, dC, dD, d_workspace, streams, num_streams))
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "cusparseLt spending time : " << ms << "ms" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);


    // memory copy device to host
    CHECK_CUDA(hipMemcpy(hA_pruned, dA, A_size, hipMemcpyDeviceToHost))   // pruned with 50sparsity matrix A copy to host 
    CHECK_CUDA(hipMemcpy(hC, dC, C_size, hipMemcpyDeviceToHost))  

    // destroy plan and handle
    CHECK_CUSPARSE(cusparseLtMatDescriptorDestroy(&matA))
    CHECK_CUSPARSE(cusparseLtMatDescriptorDestroy(&matB))
    CHECK_CUSPARSE(cusparseLtMatDescriptorDestroy(&matC))
    CHECK_CUSPARSE(cusparseLtMatmulPlanDestroy(&plan))
    CHECK_CUSPARSE(cusparseLtDestroy(&handle))
    
    // device memory deallocation
    CHECK_CUDA(hipFree(dA_compressed))
    CHECK_CUDA(hipFree(dA))
    CHECK_CUDA(hipFree(dB))
    CHECK_CUDA(hipFree(dC))
    CHECK_CUDA(hipFree(d_valid))

    return EXIT_SUCCESS;
}

template <typename T>
int cusMatmulCoo(T* hA_pruned, T* hB, T* hC, const int m, const int n, const int k)
{
    auto          order = HIPSPARSE_ORDER_ROW;
    auto          opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          type = HIP_R_32F;
    auto          compute_type = HIP_R_32F;
    bool     is_rowmajor = (order == HIPSPARSE_ORDER_ROW);
    bool     isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    bool     isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    auto     num_A_rows = (isA_transposed) ? k : m;
    auto     num_A_cols = (isA_transposed) ? m : k;
    auto     num_B_rows = (isB_transposed) ? n : k;
    auto     num_B_cols = (isB_transposed) ? k : n;
    auto     num_C_rows = m;
    auto     num_C_cols = n;
    auto     lda = (is_rowmajor) ? num_A_cols : num_A_rows;
    auto     ldb = (is_rowmajor) ? num_B_cols : num_B_rows;
    auto     ldc = (is_rowmajor) ? num_C_cols : num_C_rows;
    auto     A_height = (is_rowmajor) ? num_A_rows : num_A_cols;
    auto     B_height = (is_rowmajor) ? num_B_rows : num_B_cols;
    auto     C_height = (is_rowmajor) ? num_C_rows : num_C_cols;
    auto     A_size = A_height * lda * sizeof(T);
    auto     B_size = B_height * ldb * sizeof(T);
    auto     C_size = C_height * ldc * sizeof(T);
            
    float alpha = 1.0f;
    float beta = 0.0f;
  
    // Device memory management
    T * dA_pruned, * dB, * dC;
    CHECK_CUDA(hipMalloc((void**)&dA_pruned, A_size))
    CHECK_CUDA(hipMalloc((void**)&dB, B_size))
    CHECK_CUDA(hipMalloc((void**)&dC, C_size))
    CHECK_CUDA(hipMemcpy(dA_pruned, hA_pruned, A_size, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dB, hB, B_size, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemset(dC, 0, C_size))   
    
    int* d_coo_rows, * d_coo_columns;
    T* d_coo_values;
        
    // convert dense matrix tmpA --> sparse matrix matA in COO format
    hipsparseHandle_t     handle = NULL;
    hipsparseSpMatDescr_t matA;
    hipsparseDnMatDescr_t tmpA, matB, matC;
    void* dBuffer1 = NULL;
    void* dBuffer2 = NULL;
    size_t bufferSize1 = 0;
    size_t bufferSize2 = 0;
     
    CHECK_CUSPARSE(hipsparseCreate(&handle))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&tmpA, num_A_rows, num_A_cols, lda, dA_pruned, type, order))
    CHECK_CUSPARSE(hipsparseCreateCoo(&matA, num_A_rows, num_A_cols, 0, NULL, NULL, NULL, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, type)) // Create sparse matrix A in Coo format   
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(handle, tmpA, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize1))
    CHECK_CUDA(hipMalloc(&dBuffer1, bufferSize1))
    CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, tmpA, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer1)) // execute Dense to Sparse conversion                                                                                                                         
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matA, &num_rows_tmp, &num_cols_tmp, &nnz))    // get number of non-zero elements
    CHECK_CUDA(hipMalloc((void**)&d_coo_rows, nnz * sizeof(int)))
    CHECK_CUDA(hipMalloc((void**)&d_coo_columns, nnz * sizeof(int)))
    CHECK_CUDA(hipMalloc((void**)&d_coo_values, nnz * sizeof(T)))
    // reset row indices, column indices, and values pointers
    CHECK_CUSPARSE(hipsparseCooSetPointers(matA, d_coo_rows, d_coo_columns, d_coo_values))
    // execute Dense to Sparse conversion
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, tmpA, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer1))
            
    // -----------------------------------------------------------------------------------------
    // Create dense matrix 
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, num_B_rows, num_B_cols, ldb, dB, type, order))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, num_C_rows, num_C_cols, ldc, dC, type, order))
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, opA, opB, &alpha, matA, matB, &beta, matC, compute_type, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize2))
    CHECK_CUDA(hipMalloc(&dBuffer2, bufferSize2))
    
    // perform matrix multiplication SpMM
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    CHECK_CUSPARSE(hipsparseSpMM(handle, opA, opB, &alpha, matA, matB, &beta, matC, compute_type, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer2))
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "cusparseCOO spending time : " << ms << "ms" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);


    CHECK_CUDA(hipMemcpy(hC, dC, C_size, hipMemcpyDeviceToHost))
     
    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(tmpA))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB))
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
    CHECK_CUSPARSE(hipsparseDestroy(handle))
    // free memory
    CHECK_CUDA(hipFree(dBuffer1))
    CHECK_CUDA(hipFree(dBuffer2))
    CHECK_CUDA(hipFree(d_coo_rows))
    CHECK_CUDA(hipFree(d_coo_columns))
    CHECK_CUDA(hipFree(d_coo_values))
    CHECK_CUDA(hipFree(dA_pruned))
    CHECK_CUDA(hipFree(dB))
    CHECK_CUDA(hipFree(dC))

    return EXIT_SUCCESS;
}

template <typename T>
int cusMatmulCsr(T* hA_pruned, T* hB, T* hC, const int m, const int n, const int k)
{
    auto          order = HIPSPARSE_ORDER_ROW;
    auto          opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          type = HIP_R_32F;
    auto          compute_type = HIP_R_32F;
    bool     is_rowmajor = (order == HIPSPARSE_ORDER_ROW);
    bool     isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    bool     isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    auto     num_A_rows = (isA_transposed) ? k : m;
    auto     num_A_cols = (isA_transposed) ? m : k;
    auto     num_B_rows = (isB_transposed) ? n : k;
    auto     num_B_cols = (isB_transposed) ? k : n;
    auto     num_C_rows = m;
    auto     num_C_cols = n;
    auto     lda = (is_rowmajor) ? num_A_cols : num_A_rows;
    auto     ldb = (is_rowmajor) ? num_B_cols : num_B_rows;
    auto     ldc = (is_rowmajor) ? num_C_cols : num_C_rows;
    auto     A_height = (is_rowmajor) ? num_A_rows : num_A_cols;
    auto     B_height = (is_rowmajor) ? num_B_rows : num_B_cols;
    auto     C_height = (is_rowmajor) ? num_C_rows : num_C_cols;
    auto     A_size = A_height * lda * sizeof(T);
    auto     B_size = B_height * ldb * sizeof(T);
    auto     C_size = C_height * ldc * sizeof(T);
      
    float alpha = 1.0f;
    float beta = 0.0f;

    // Device memory management
    T* dA_pruned, * dB, * dC;
    CHECK_CUDA(hipMalloc((void**)&dA_pruned, A_size))
    CHECK_CUDA(hipMalloc((void**)&dB, B_size))
    CHECK_CUDA(hipMalloc((void**)&dC, C_size))
    CHECK_CUDA(hipMemcpy(dA_pruned, hA_pruned, A_size, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dB, hB, B_size, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemset(dC, 0, C_size))

    int* d_csr_offsets, * d_csr_columns;
    T* d_csr_values;
    CHECK_CUDA(hipMalloc((void**)&d_csr_offsets, (num_A_rows + 1) * sizeof(int)))

    // convert dense matrix tmpA --> sparse matrix matA in CSR format
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t tmpA, matB, matC;
    hipsparseSpMatDescr_t matA;
    void* dBuffer1 = NULL;
    void* dBuffer2 = NULL;
    size_t               bufferSize1 = 0;
    size_t               bufferSize2 = 0;

    CHECK_CUSPARSE(hipsparseCreate(&handle))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&tmpA, num_A_rows, num_A_cols, lda, dA_pruned, type, order))
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, num_A_rows, num_A_cols, 0, d_csr_offsets, NULL, NULL, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, type)) // Create sparse matrix A in Csr format   
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(handle, tmpA, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize1))
    CHECK_CUDA(hipMalloc(&dBuffer1, bufferSize1))
    CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, tmpA, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer1)) // execute Dense to Sparse conversion                                                                                                                         
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matA, &num_rows_tmp, &num_cols_tmp, &nnz))    // get number of non-zero elements   
    CHECK_CUDA(hipMalloc((void**)&d_csr_columns, nnz * sizeof(int)))
    CHECK_CUDA(hipMalloc((void**)&d_csr_values, nnz * sizeof(T)))
    // reset row indices, column indices, and values pointers
    CHECK_CUSPARSE(hipsparseCsrSetPointers(matA, d_csr_offsets, d_csr_columns, d_csr_values))
    // execute Dense to Sparse conversion
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, tmpA, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer1))
    // -----------------------------------------------------------------------------------------
    // Create dense matrix 
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, num_B_rows, num_B_cols, ldb, dB, type, order))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, num_C_rows, num_C_cols, ldc, dC, type, order))
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, opA, opB, &alpha, matA, matB, &beta, matC, compute_type, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize2))
    CHECK_CUDA(hipMalloc(&dBuffer2, bufferSize2))

    // execute SpMM
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    CHECK_CUSPARSE(hipsparseSpMM(handle, opA, opB, &alpha, matA, matB, &beta, matC, compute_type, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer2))
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "cusparseCSR spending time : " << ms << "ms" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipMemcpy(hC, dC, C_size, hipMemcpyDeviceToHost))

    
 /*   T* v = (T*)malloc(sizeof(T) * nnz);
    int* o = (int*)malloc(sizeof(int) * (num_A_rows + 1));
    int* c = (int*)malloc(sizeof(int) * nnz);
    hipMemcpy(v, d_csr_values, sizeof(int) * nnz, hipMemcpyDeviceToHost);
    hipMemcpy(o, d_csr_offsets, sizeof(int) * (num_A_rows + 1), hipMemcpyDeviceToHost);
    hipMemcpy(c, d_csr_columns, sizeof(int) * nnz, hipMemcpyDeviceToHost);
    

    std::cout << "\nnnz -->\n";
    for (int i = 0; i < nnz; i++) {
        std::cout << v[i] << " ";
    }
    std::cout << std::endl;  std::cout << "\no -->";
    for (int i = 0; i < num_A_rows + 1; i++) {
        std::printf("%d ", o[i]);
    }
    std::cout << std::endl;  std::cout << "\nc -->";
    for (int i = 0; i < nnz; i++) {
        std::printf("%d ", c[i]);
    }
    
    free(v);
    free(o);
    free(c);*/


    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(tmpA))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
    CHECK_CUSPARSE(hipsparseDestroy(handle))

    // free memory
    CHECK_CUDA(hipFree(dBuffer1))
    CHECK_CUDA(hipFree(dBuffer2))
    CHECK_CUDA(hipFree(d_csr_offsets))
    CHECK_CUDA(hipFree(d_csr_columns))
    CHECK_CUDA(hipFree(d_csr_values))
    CHECK_CUDA(hipFree(dB))
    CHECK_CUDA(hipFree(dC))

    return EXIT_SUCCESS;
}

template <typename T>
int cusMatmulCsc(T* hA_pruned, T* hB, T* hC, const int m, const int n, const int k)
{
    auto          orderA = HIPSPARSE_ORDER_COL;
    auto          orderB = HIPSPARSE_ORDER_ROW;
    auto          orderC = HIPSPARSE_ORDER_ROW;
    auto          opA = HIPSPARSE_OPERATION_TRANSPOSE;
    auto          opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    auto          type = HIP_R_32F;
    auto          compute_type = HIP_R_32F;
    bool     is_rowmajorA = (orderA == HIPSPARSE_ORDER_ROW);
    bool     is_rowmajorB = (orderB == HIPSPARSE_ORDER_ROW);
    bool     is_rowmajorC = (orderC == HIPSPARSE_ORDER_ROW);
    bool     isA_transposed = (opA != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    bool     isB_transposed = (opB != HIPSPARSE_OPERATION_NON_TRANSPOSE);
    auto     num_A_rows = (isA_transposed) ? k : m;
    auto     num_A_cols = (isA_transposed) ? m : k;
    auto     num_B_rows = (isB_transposed) ? n : k;
    auto     num_B_cols = (isB_transposed) ? k : n;
    auto     num_C_rows = m;
    auto     num_C_cols = n;
    auto     lda = (is_rowmajorA) ? num_A_cols : num_A_rows;
    auto     ldb = (is_rowmajorB) ? num_B_cols : num_B_rows;
    auto     ldc = (is_rowmajorC) ? num_C_cols : num_C_rows;
    auto     A_height = (is_rowmajorA) ? num_A_rows : num_A_cols;
    auto     B_height = (is_rowmajorB) ? num_B_rows : num_B_cols;
    auto     C_height = (is_rowmajorC) ? num_C_rows : num_C_cols;
    auto     A_size = A_height * lda * sizeof(T);
    auto     B_size = B_height * ldb * sizeof(T);
    auto     C_size = C_height * ldc * sizeof(T);

    float alpha = 1.0f;
    float beta = 0.0f;

    // Device memory management
    T* dA_pruned, * dB, * dC;
    CHECK_CUDA(hipMalloc((void**)&dA_pruned, A_size))
    CHECK_CUDA(hipMalloc((void**)&dB, B_size))
    CHECK_CUDA(hipMalloc((void**)&dC, C_size))
    CHECK_CUDA(hipMemcpy(dA_pruned, hA_pruned, A_size, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemcpy(dB, hB, B_size, hipMemcpyHostToDevice))
    CHECK_CUDA(hipMemset(dC, 0, C_size))

    int* d_csc_offsets, * d_csc_rows;
    T* d_csc_values;
    CHECK_CUDA(hipMalloc((void**)&d_csc_offsets, (num_A_cols + 1) * sizeof(int)))

    // convert dense matrix tmpA --> sparse matrix matA in CSC format
    hipsparseHandle_t     handle = NULL;
    hipsparseDnMatDescr_t tmpA, matB, matC;
    hipsparseSpMatDescr_t matA;
    void* dBuffer1 = NULL;
    void* dBuffer2 = NULL;
    size_t               bufferSize1 = 0;
    size_t               bufferSize2 = 0;

    CHECK_CUSPARSE(hipsparseCreate(&handle))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&tmpA, num_A_rows, num_A_cols, lda, dA_pruned, type, orderA))
    CHECK_CUSPARSE(hipsparseCreateCsc(&matA, num_A_rows, num_A_cols, 0, d_csc_offsets, NULL, NULL, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, type)) // Create sparse matrix A in Coo format   
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(handle, tmpA, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, &bufferSize1))
    CHECK_CUDA(hipMalloc(&dBuffer1, bufferSize1))
    CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle, tmpA, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer1)) // execute Dense to Sparse conversion                                                                                                                         
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matA, &num_rows_tmp, &num_cols_tmp, &nnz))    // get number of non-zero elements  
    CHECK_CUDA(hipMalloc((void**)&d_csc_rows, nnz * sizeof(int)))
    CHECK_CUDA(hipMalloc((void**)&d_csc_values, nnz * sizeof(T)))
    // reset row indices, column indices, and values pointers
    CHECK_CUSPARSE(hipsparseCscSetPointers(matA, d_csc_offsets, d_csc_rows, d_csc_values))
    // execute Dense to Sparse conversion
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle, tmpA, matA, HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT, dBuffer1))
    // -----------------------------------------------------------------------------------------
    // Create dense matrix 
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, num_B_rows, num_B_cols, ldb, dB, type, orderB))
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, num_C_rows, num_C_cols, ldc, dC, type, orderC))
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(handle, opA, opB, &alpha, matA, matB, &beta, matC, compute_type, HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize2))
    CHECK_CUDA(hipMalloc(&dBuffer2, bufferSize2))

    // execute SpMM
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    CHECK_CUSPARSE(hipsparseSpMM(handle, opA, opB, &alpha, matA, matB, &beta, matC, compute_type, HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer2))
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "cusparseCSC spending time : " << ms << "ms" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    CHECK_CUDA(hipMemcpy(hC, dC, C_size, hipMemcpyDeviceToHost))

    
    //T* v = (T*)malloc(sizeof(T) * nnz);
    //int* o = (int*)malloc(sizeof(int) * (num_A_cols + 1));
    //int* r = (int*)malloc(sizeof(int) * nnz);
    //hipMemcpy(v, d_csc_values, sizeof(int) * nnz, hipMemcpyDeviceToHost);
    //hipMemcpy(o, d_csc_offsets, sizeof(int) * (num_A_cols + 1), hipMemcpyDeviceToHost);
    //hipMemcpy(r, d_csc_rows, sizeof(int) * nnz, hipMemcpyDeviceToHost);
    //

    //std::cout << "\nnnz -->\n";
    //for (int i = 0; i < nnz; i++) {
    //    std::cout << v[i] << " ";
    //}
    //std::cout << std::endl; std::cout << "\no -->\n";
    //for (int i = 0; i < num_A_cols + 1; i++) {
    //    std::printf("%d ", o[i]);
    //}
    //std::cout << std::endl; std::cout << "\nr -->\n";
    //for (int i = 0; i < nnz; i++) {
    //    std::printf("%d ",r[i]);
    //}

    //free(v);
    //free(o);
    //free(r);
    

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(tmpA))
    CHECK_CUSPARSE(hipsparseDestroySpMat(matA))
    CHECK_CUSPARSE(hipsparseDestroy(handle))

    // free memory
    CHECK_CUDA(hipFree(dBuffer1))
    CHECK_CUDA(hipFree(dBuffer2))
    CHECK_CUDA(hipFree(d_csc_offsets))
    CHECK_CUDA(hipFree(d_csc_rows))
    CHECK_CUDA(hipFree(d_csc_values))
    CHECK_CUDA(hipFree(dB))
    CHECK_CUDA(hipFree(dC))

    return EXIT_SUCCESS;
}